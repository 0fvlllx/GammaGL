#include "hip/hip_runtime.h"
#include "segment_max_cuda.h"
#include <torch/torch.h>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <torch/script.h>
#include <ATen/cuda/HIPContext.h>
#include <assert.h>

using torch::autograd::AutogradContext;
using torch::autograd::Variable;
using torch::autograd::variable_list;

#define THREADS 1024
#define BLOCKS(N) (N + THREADS - 1) / THREADS

inline __device__ void atomic_max_float(float *addr, float value) {
  int *addr_as_i = (int *)addr;
  int old = *addr_as_i;
  int assumed;
  do{
    assumed = old;
    old = atomicCAS(addr_as_i, assumed,
                    __float_as_int(max(value, __int_as_float(assumed))));
  } while (assumed != old);
}

template <typename scalar_t>
__global__ void segment_max_cuda_forward_kernel(const scalar_t *src_data, const int64_t *index_data,
                               scalar_t *out_data, int E, int K, int N, int numel) {
  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int e = (thread_idx / K) % E;
  int k = thread_idx % K;
  if (thread_idx < numel)  {
    // TODO: support more data type
    int idx = index_data[e];
    atomic_max_float(out_data + idx * K + k,
                     src_data[thread_idx]);
  }
}

// TODO: fuse segment & arg_segment to one kernel function.
template <typename scalar_t>
__global__ void
arg_segment_max_cuda_forward_kernel(const scalar_t *src_data, const int64_t *index_data,
                   scalar_t *out_data, int64_t *arg_out_data, int E,
                   int K, int N, int numel) {
  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int e = (thread_idx / K) % E;
  int k = thread_idx % K;

  if (thread_idx < numel) {
    int idx = index_data[e];
    if (src_data[thread_idx] == out_data[idx * K + k]) {
      arg_out_data[idx * K + k] = e;
    }
  }
}

std::tuple<torch::Tensor, torch::Tensor>
segment_max_cuda_forward(torch::Tensor src, torch::Tensor index, int64_t N) {
  // check inputs
  TORCH_CHECK(src.device().is_cuda(), "src must be CUDA tensor");
  TORCH_CHECK(index.device().is_cuda(), "index must be CUDA tensor");
  TORCH_CHECK_INDEX(src.dim() == 2, "src dimension should be 2, but got ", src.dim());
  TORCH_CHECK_INDEX(index.dim() == 1, "index dimension should be 1, but got ", index.dim());
  TORCH_CHECK_INDEX(src.size(0) == index.size(0), "fisrt dimension of src and index should be same");
  // only support float Tensor
  TORCH_CHECK_TYPE(src.scalar_type() == c10::ScalarType::Float, "src should be float Tensor")
  hipSetDevice(src.get_device());
  src = src.contiguous();

  auto sizes = src.sizes().vec();
  sizes[0] = N > *index.max().cpu().data_ptr<int64_t>()
                 ? N
                 : *index.max().cpu().data_ptr<int64_t>();
  torch::Tensor out = torch::empty(sizes, src.options());
  // TORCH_CHECK(out.device().is_cuda(), "out must be CUDA tensor");
  torch::Tensor arg_out = torch::full_like(out, 0, index.options());
  int64_t *arg_out_data = arg_out.data_ptr<int64_t>();
  if (src.numel() == 0) {
    out.fill_(0);
    return std::make_tuple(out, arg_out);
  }

  out.fill_(std::numeric_limits<int64_t>::lowest());
  auto E = src.size(0);
  auto K = src.size(1);
  auto stream = at::cuda::getCurrentCUDAStream();

  // AT_DISPATCH_ALL_TYPES(src.scalar_type(), "__ops_name",  [&] {
  using scalar_t = float; // temporary usage, delete later
  auto src_data = src.data_ptr<scalar_t>();
  auto out_data = out.data_ptr<scalar_t>();
  auto index_data = index.data_ptr<int64_t>();

  segment_max_cuda_forward_kernel<scalar_t>
      <<<BLOCKS(src.numel()), THREADS, 0, stream>>>(
          src_data, index_data, out_data, E, K, N, src.numel());

  out.masked_fill_(out == std::numeric_limits<int64_t>::lowest(), (scalar_t)0);

  arg_segment_max_cuda_forward_kernel<scalar_t>
      <<<BLOCKS(src.numel()), THREADS, 0, stream>>>(
          src_data, index_data, out_data, arg_out_data, E, K, N,
          src.numel());
  // });

  return std::make_tuple(out, arg_out);
}
